#include <cstdio>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition using unified memory
__global__ void vectorAddUnified(int *c, int *a, int *b, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 100;
    size_t size = N * sizeof(int);

    // Unified memory allocation
    int *a, *b, *c;
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize host arrays with values
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;  // Ensure N elements are processed
    vectorAddUnified<<<numBlocks, blockSize>>>(c, a, b, N);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Print results
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
