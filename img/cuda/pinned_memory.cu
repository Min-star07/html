#include <cstdio>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition using pinned memory
__global__ void vectorAddPinned(int *c, int *a, int *b, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 100;
    size_t size = N * sizeof(int);

    // Allocate pinned host memory
    int *h_a, *h_b, *h_c;
    hipHostAlloc((void**)&h_a, size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_b, size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_c, size, hipHostMallocDefault);

    // Initialize host arrays with values
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device memory pointers
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from pinned host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;  // Ensure N elements are processed
    vectorAddPinned<<<numBlocks, blockSize>>>(d_c, d_a, d_b, N);
    
    // Copy result from device to pinned host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free pinned host memory
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}
